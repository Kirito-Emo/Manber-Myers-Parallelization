// Created by Emanuele (https://github.com/Kirito-Emo)

#include "cuda_malloc_utils.h"
#include "cuda_utils.h"
#include <hip/hip_runtime_api.h>

// Initializes a custom asynchronous memory pool for reuse across allocations
void cuda_mem_pool_init(hipMemPool_t* pool)
{
    int device;
    CUDA_CHECK(hipGetDevice(&device));

    // Create a new memory pool instead of using the default one
    hipMemPoolProps props = {};
    props.allocType = hipMemAllocationTypePinned;
    props.handleTypes = hipMemHandleTypeNone;
    props.location.type = hipMemLocationTypeDevice;
    props.location.id = device;

    CUDA_CHECK(hipMemPoolCreate(pool, &props));

    // Set optional attributes
    size_t threshold = 1ULL << 32; // 4 GB threshold before releasing memory
    CUDA_CHECK(hipMemPoolSetAttribute(*pool, hipMemPoolAttrReleaseThreshold, &threshold));

    // Set the pool as active for the current device (optional)
    CUDA_CHECK(hipDeviceSetMemPool(device, *pool));
}

// Allocates memory from the given pool
void* cuda_malloc_async(hipMemPool_t pool, size_t size, hipStream_t stream)
{
    void* ptr = nullptr;
    CUDA_CHECK(hipMallocFromPoolAsync(&ptr, size, pool, stream));
    return ptr;
}

// Frees memory asynchronously
void cuda_free_async(void* ptr, hipStream_t stream)
{
    CUDA_CHECK(hipFreeAsync(ptr, stream));
}

// Destroys the memory pool if custom
void cuda_mem_pool_destroy(hipMemPool_t pool)
{
    // Only destroy if the pool is not the default one
    CUDA_CHECK(hipMemPoolDestroy(pool));
}
